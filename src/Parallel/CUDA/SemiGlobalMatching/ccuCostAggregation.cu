#include "../../../../include/Common/cmTypeDefs.h"
#include "../../../../include/Parallel/CUDA/ccuDeclarations.h"
namespace Parallel {
	namespace CUDA {
		namespace SemiGlobalMatching {
			namespace CostAggregator {
				cu_global void cusmParallelCostAggregationLR(CU_ARG u8* imageData, CU_ARG u32* costMatrix, u32 imageWidth, u32 imageHeight, i32 minDisparity, u32 disparityRange, CU_ARG OUT_ARG u32* refinedMatrix, u8 direction, u8 threads, CU_ARG u32* optCostR) {
					f64 p1 = 10,p2=150;
					u32 lastMin = U32_MAX;
					i32 parallel = threadIdx.x;
					u32* optCost = optCostR + (2 * disparityRange * parallel);
					for (u32 j = parallel; j < imageHeight; j+=threads) {
						lastMin = U32_MAX;
						i32 startCoord = direction ? imageWidth - 1 : 0;
						i32 stopCoord = direction ? -1 : imageWidth;
						i32 deltaCoord = direction ? -1 : 1;
						for (u32 k = 0; k < disparityRange * 2; k++) {
							optCost[k] = U32_MAX;
						}
						for (u32 k = 0; k < disparityRange; k++) {
							get_pixel(optCost, k, startCoord & 1, disparityRange, 2) = get_pixel3(costMatrix, startCoord, j, k, imageWidth, imageHeight, disparityRange);
							lastMin = Min(lastMin, get_pixel(optCost, k, startCoord & 1, disparityRange, 2));
							get_pixel3(refinedMatrix, startCoord, j, k, imageWidth, imageHeight, disparityRange) += get_pixel3(costMatrix, startCoord, j, k, imageWidth, imageHeight, disparityRange) / 8;
						}
						for (i32 k = startCoord + deltaCoord; k != stopCoord; k += deltaCoord) {
							i32 newMin = I32_MAX;
							for (u32 d = 0; d < disparityRange; d++) {
								get_pixel(optCost, d, k & 1, disparityRange, 2) = get_pixel3(costMatrix, k, j, d, imageWidth, imageHeight, disparityRange);
								i32 addedValue = get_pixel(optCost, d, (k + 1) & 1, disparityRange, 2);
								if (d > 0) {
									if ((i32)(get_pixel(optCost, d - 1, (k + 1) & 1, disparityRange, 2) + p1) < addedValue) {
										addedValue = get_pixel(optCost, d - 1, (k + 1) & 1, disparityRange, 2) + p1;
									}
								}
								if (d < disparityRange - 1) {
									if ((i32)(get_pixel(optCost, d + 1, (k + 1) & 1, disparityRange, 2) + p1) < addedValue) {
										addedValue = (get_pixel(optCost, d + 1, (k + 1) & 1, disparityRange, 2) + p1);
									}
								}
								i32 p2Coef = p2 / (Abs((i32)get_pixel(imageData, k, j, imageWidth, imageHeight) - (i32)get_pixel(imageData, k - deltaCoord, j, imageWidth, imageHeight)) + 1);
								p2Coef = Max(p2Coef, (i32)p1);
								if ((i32)(lastMin + p2Coef) < addedValue) {
									addedValue = Min(addedValue, (i32)(lastMin + p2Coef));
								}

								get_pixel(optCost, d, k & 1, disparityRange, 2) += (addedValue - lastMin);
								newMin = Min(newMin, (i32)get_pixel(optCost, d, k & 1, disparityRange, 2));
								get_pixel3(refinedMatrix, k, j, d, imageWidth, imageHeight, disparityRange) += get_pixel(optCost, d, k & 1, disparityRange, 2) / 8;
							}
							lastMin = newMin;
						}
					}
				}
				cu_global void cusmParallelCostAggregationUD(CU_ARG u8* imageData, CU_ARG u32* costMatrix, u32 imageWidth, u32 imageHeight, i32 minDisparity, u32 disparityRange, CU_ARG OUT_ARG u32* refinedMatrix, u8 direction, u8 threads, CU_ARG u32* optCostR) {
					f64 p1 = 10,p2=150;
					u32 lastMin = U32_MAX;
					i32 parallel = threadIdx.x;
					u32* optCost = optCostR + (2 * disparityRange * parallel);
					for (u32 i = parallel; i < imageWidth; i+=threads) {
						lastMin = U32_MAX;
						//Current pixel is (?,j)
						i32 startCoord = direction ? imageHeight - 1 : 0;
						i32 stopCoord = direction ? -1 : imageHeight;
						i32 deltaCoord = direction ? -1 : 1;
						for (u32 k = 0; k < disparityRange * 2; k++) {
							optCost[k] = U32_MAX;
						}
						//Init Cond for Dynamic Programming
						for (u32 k = 0; k < disparityRange; k++) {
							lastMin = Min(lastMin, get_pixel(optCost, k, startCoord & 1, disparityRange, 2) = get_pixel3(costMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange));
							get_pixel3(refinedMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange) += get_pixel3(costMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange) / 8;
						}
						//Status Updating
						for (i32 k = startCoord + deltaCoord; k != stopCoord; k += deltaCoord) {
							//Updating f[k,j,d] from f[k',j,d']
							i32 newMin = I32_MAX;
							for (u32 d = 0; d < disparityRange; d++) {
								get_pixel(optCost, d, k & 1, disparityRange, 2) = get_pixel3(costMatrix, i, k, d, imageWidth, imageHeight, disparityRange);
								i32 addedValue = get_pixel(optCost, d, (k + 1) & 1, disparityRange, 2);
								if (d > 0) {
									addedValue = Min(addedValue, (i32)(get_pixel(optCost, d - 1, (k + 1) & 1, disparityRange, 2) + p1));
								}
								if (d < disparityRange - 1u) {
									addedValue = Min(addedValue, (i32)(get_pixel(optCost, d + 1, (k + 1) & 1, disparityRange, 2) + p1));
								}
								i32 p2Coef = p2 / (Abs((i32)get_pixel(imageData, i, k, imageWidth, imageHeight) - (i32)get_pixel(imageData, i, k - deltaCoord, imageWidth, imageHeight)) + 1);
								p2Coef = Max(p2Coef, (i32)p1);
								addedValue = Min(addedValue, (i32)(lastMin + p2Coef));
								get_pixel(optCost, d, k & 1, disparityRange, 2) += (addedValue - lastMin);
								newMin = Min(newMin, (i32)get_pixel(optCost, d, k & 1, disparityRange, 2));
								get_pixel3(refinedMatrix, i, k, d, imageWidth, imageHeight, disparityRange) += get_pixel(optCost, d, k & 1, disparityRange, 2) / 8;
							}
							lastMin = newMin;
						}
					}
				}
				cu_global void cusmParallelCostAggregationND(CU_ARG u8* imageData, CU_ARG u32* costMatrix, u32 imageWidth, u32 imageHeight, i32 minDisparity, u32 disparityRange, CU_ARG OUT_ARG u32* refinedMatrix, u8 direction, u8 threads, CU_ARG u32* optCostR) {
					f64 p1 = 10,p2=150;
					u32 lastMin = U32_MAX;
					i32 parallel = threadIdx.x;
					u32* optCost = optCostR + (2 * disparityRange * parallel);
					for (u32 i = parallel; i < imageWidth; i+=threads) {
						lastMin = U32_MAX;
						//Current pixel is (?,j)
						i32 startCoord = direction ? imageHeight - 1 : 0;
						i32 stopCoord = direction ? -1 : imageHeight;
						i32 deltaCoord = direction ? -1 : 1;
						for (u32 k = 0; k < disparityRange * 2; k++) {
							optCost[k] = U32_MAX;
						}
						//Init Cond for Dynamic Programming
						for (u32 k = 0; k < disparityRange; k++) {
							lastMin = Min(lastMin, (i32)(get_pixel(optCost, k, startCoord & 1, disparityRange, 2) = get_pixel3(costMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange)));
							get_pixel3(refinedMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange) += get_pixel3(costMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange) / 8;
						}
						//Status Updating
						// k - vertical
						// q - horizontal
						for (i32 k = startCoord + deltaCoord, q = i; k != stopCoord; k += deltaCoord) {
							i32 newMin = I32_MAX;
							q -= deltaCoord;
							q += (i32)imageWidth;
							q %= (i32)imageWidth;
							for (u32 d = 0; d < disparityRange; d++) {
								get_pixel(optCost, d, k & 1, disparityRange, 2) = get_pixel3(costMatrix, q, k, d, imageWidth, imageHeight, disparityRange);
								i32 addedValue = get_pixel(optCost, d, (k + 1) & 1, disparityRange, 2);
								if (d > 0) {
									addedValue = Min(addedValue, (i32)(get_pixel(optCost, d - 1, (k + 1) & 1, disparityRange, 2) + p1));
								}
								if (d < disparityRange - 1u) {
									addedValue = Min(addedValue, (i32)(get_pixel(optCost, d + 1, (k + 1) & 1, disparityRange, 2) + p1));
								}
								i32 p2Coef = p2 / (Abs((i32)get_pixel(imageData, q, k, imageWidth, imageHeight) - (i32)get_pixel(imageData, (q + deltaCoord + (i32)imageWidth) % (i32)imageWidth, k - deltaCoord, imageWidth, imageHeight)) + 1);
								p2Coef = Max(p2Coef, (i32)p1);
								addedValue = Min(addedValue, (i32)(lastMin + p2Coef));
								get_pixel(optCost, d, k & 1, disparityRange, 2) += (addedValue - lastMin);
								newMin = Min(newMin, (i32)get_pixel(optCost, d, k & 1, disparityRange, 2));
								get_pixel3(refinedMatrix, q, k, d, imageWidth, imageHeight, disparityRange) += get_pixel(optCost, d, k & 1, disparityRange, 2) / 8;
							}
							lastMin = newMin;
						}
					}
				}
				cu_global void cusmParallelCostAggregationPD(CU_ARG u8* imageData, CU_ARG u32* costMatrix, u32 imageWidth, u32 imageHeight, i32 minDisparity, u32 disparityRange, CU_ARG OUT_ARG u32* refinedMatrix, u8 direction, u8 threads, CU_ARG u32* optCostR) {
					//printf("I am In UD\n");
					f64 p1 = 10,p2=150;
					u32 lastMin = U32_MAX;
					i32 parallel = threadIdx.x;
					u32* optCost = optCostR + (2 * disparityRange * parallel);
					for (u32 i = parallel; i < imageWidth; i+=threads) {
						lastMin = U32_MAX;
						i32 startCoord = direction ? imageHeight - 1 : 0;
						i32 stopCoord = direction ? -1 : imageHeight;
						i32 deltaCoord = direction ? -1 : 1;
						for (u32 k = 0; k < disparityRange * 2; k++) {
							optCost[k] = U32_MAX;
						}
						for (u32 k = 0; k < disparityRange; k++) {
							get_pixel(optCost, k, startCoord & 1, disparityRange, 2) = get_pixel3(costMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange);
							lastMin = Min(lastMin, (i32)get_pixel(optCost, k, startCoord & 1, disparityRange, 2));
							get_pixel3(refinedMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange) += get_pixel3(costMatrix, i, startCoord, k, imageWidth, imageHeight, disparityRange) / 8;
						}
						for (i32 k = startCoord + deltaCoord, q = i; k != stopCoord; k += deltaCoord) {
							i32 newMin = I32_MAX;
							q += deltaCoord;
							q += (i32)imageWidth;
							q %= (i32)imageWidth;
							for (u32 d = 0; d < disparityRange; d++) {
								get_pixel(optCost, d, k & 1, disparityRange, 2) = get_pixel3(costMatrix, q, k, d, imageWidth, imageHeight, disparityRange);
								i32 addedValue = get_pixel(optCost, d, (k + 1) & 1, disparityRange, 2);
								if (d > 0) {
									addedValue = Min(addedValue, (i32)(get_pixel(optCost, d - 1, (k + 1) & 1, disparityRange, 2) + p1));
								}
								if (d < disparityRange - 1u) {
									addedValue = Min(addedValue, (i32)(get_pixel(optCost, d + 1, (k + 1) & 1, disparityRange, 2) + p1));
								}
								i32 p2Coef = p2 / (Abs((i32)get_pixel(imageData, q, k, imageWidth, imageHeight) - (i32)get_pixel(imageData, (q - deltaCoord + (i32)imageWidth) % (i32)imageWidth, k - deltaCoord, imageWidth, imageHeight)) + 1);
								p2Coef = Max(p2Coef, (i32)p1);
								addedValue = Min(addedValue, (i32)(lastMin + p2Coef));
								get_pixel(optCost, d, k & 1, disparityRange, 2) += (addedValue - lastMin);
								newMin = Min(newMin, (i32)get_pixel(optCost, d, k & 1, disparityRange, 2));
								get_pixel3(refinedMatrix, q, k, d, imageWidth, imageHeight, disparityRange) += get_pixel(optCost, d, k & 1, disparityRange, 2) / 8;
							}
							lastMin = newMin;
						}
					}
				}
				void cusmParallelCostAggregationFourPathCaller(u8* imageData, u32* costMatrix, u32 imageWidth, u32 imageHeight, i32 minDisparity, u32 disparityRange, OUT_ARG u32* refinedMatrix, u8 direction, u8 threads, u32* optCost) {
					//Copy Memory to GPU
					u8* imageDataCu = nullptr;
					u32* costMatrixCu = nullptr;
					u32* refinedMatrixCu = nullptr;
					u32* optCostF = new u32[disparityRange * 2];
					set_zero(optCostF, sizeof(u32) * 2 * disparityRange);
					u32* optCostCu = nullptr;
					hipMalloc((void**)&imageDataCu, sizeof(u8) * (usize)imageWidth * imageHeight);
					hipMemcpy(imageDataCu, imageData, sizeof(u8) * (usize)imageWidth * imageHeight, hipMemcpyHostToDevice);
					hipMalloc((void**)&costMatrixCu, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange);
					hipMemcpy(costMatrixCu, costMatrix, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange, hipMemcpyHostToDevice);
					hipMalloc((void**)&refinedMatrixCu, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange);
					hipMalloc((void**)&optCostCu, sizeof(u32) * (usize)disparityRange * 2);
					//Set Zero
					hipMemcpy(refinedMatrixCu, refinedMatrix, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange, hipMemcpyHostToDevice);
					hipMemcpy(optCostCu, optCostF, sizeof(u32) * (usize)disparityRange * 2, hipMemcpyHostToDevice);

					//Then Cost Aggregation Starts
					//TODO: Bug Fix
					/*
					cusmParallelCostAggregationLR << <1, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 0, threads, optCostCu);
					cu_sync();
					cusmParallelCostAggregationLR << <1, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 1, threads, optCostCu);
					cu_sync();
					cusmParallelCostAggregationUD << <1, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 0, threads, optCostCu);
					cu_sync();
					cusmParallelCostAggregationUD << <1, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 1, threads, optCostCu);
					cu_sync();*/
					
					//Return Value
					hipMemcpy(refinedMatrix, refinedMatrixCu, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange, hipMemcpyDeviceToHost);
					//Free GPU
					hipFree(imageDataCu);
					hipFree(refinedMatrixCu);
					hipFree(costMatrixCu);
					hipFree(optCostCu);
				}
				void cusmParallelCostAggregationEightPathCaller(u8* imageData, u32* costMatrix, u32 imageWidth, u32 imageHeight, i32 minDisparity, u32 disparityRange, OUT_ARG u32* refinedMatrix, u8 direction, u8 threads, u32* optCost) {
					//Copy Memory to GPU
					
					u8* imageDataCu = nullptr;
					u32* costMatrixCu = nullptr;
					u32* refinedMatrixCu = nullptr;
					u32* optCostCu = nullptr;
					hipMalloc((void**)&imageDataCu, sizeof(u8) * (usize)imageWidth * imageHeight);
					hipMemcpy(imageDataCu, imageData, sizeof(u8) * (usize)imageWidth * imageHeight, hipMemcpyHostToDevice);
					hipMalloc((void**)&costMatrixCu, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange);
					hipMemcpy(costMatrixCu, costMatrix, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange, hipMemcpyHostToDevice);
					hipMalloc((void**)&refinedMatrixCu, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange);
					hipMalloc((void**)&(optCostCu), sizeof(u32) * (usize)disparityRange * 2 * threads );
					usize offset = sizeof(u32) * (usize)disparityRange * 2 * threads;
					//Set Zero
					hipMemcpy(refinedMatrixCu, refinedMatrix, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange, hipMemcpyHostToDevice);
					printf("STARTS");
					//Then Cost Aggregation Starts
					cusmParallelCostAggregationLR << <1, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 0, threads, optCostCu);
					cusmParallelCostAggregationLR << <2, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 1, threads, optCostCu + offset * 1);
					cusmParallelCostAggregationUD << <3, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 0, threads, optCostCu + offset * 2);
					cusmParallelCostAggregationUD << <4, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 1, threads, optCostCu + offset * 3);
					cusmParallelCostAggregationND << <5, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 0, threads, optCostCu + offset * 4);
					cusmParallelCostAggregationND << <6, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 1, threads, optCostCu + offset * 5);
					cusmParallelCostAggregationPD << <7, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 0, threads, optCostCu + offset * 6);
					cusmParallelCostAggregationPD << <8, threads >> > (imageDataCu, costMatrixCu, imageWidth, imageHeight, minDisparity, disparityRange, refinedMatrixCu, 1, threads, optCostCu + offset * 7);
					cu_sync();
					printf("ENDS");
					//Return Value
					hipMemcpy(refinedMatrix, refinedMatrixCu, sizeof(u32) * (usize)imageWidth * imageHeight * disparityRange, hipMemcpyDeviceToHost);
					printf("%d ", get_pixel3(refinedMatrix, 14, 14, 0, imageWidth, imageHeight, disparityRange));
					printf("%d ", get_pixel3(refinedMatrix, 14, 14, 63, imageWidth, imageHeight, disparityRange));
					printf("%d ", get_pixel3(refinedMatrix, 14, 14, 32, imageWidth, imageHeight, disparityRange));

					//Free GPU
					hipFree(imageDataCu);
					hipFree(refinedMatrixCu);
					hipFree(costMatrixCu);
					hipFree(optCostCu);
				}
			}
		}
	}
}